
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <algorithm>
#include <iterator>
#include <stdlib.h>
#include <math.h>
#include <string>
#include <vector>
#include <map>
using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

typedef struct
{
	int id;
	double x;
	double y;
} City;
typedef struct
{
	double cost;
	vector<int> path;
} PathCost;

void printDistanceMatrix(float*h_distances, int numCities, int numFeatures);
double fRand(double fMin, double fMax);
vector<City> generateCities(int numCities, int gridDimX, int gridDimY);
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true);
void genKey(vector<int> set, int z, long long &key);

__global__ void computeDistances(int numInstances, int numAttributes, float* dataset, float* distances)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	int row = tid / numInstances; // instance1Index
	int column = tid - ((tid / numInstances) * numInstances); //instance2Index
	if ((tid < numInstances * numInstances))
	{
		float sum = 0;
		int instance1 = row * numAttributes;
		int instance2 = column * numAttributes;
		for (int atIdx = 1; atIdx < numAttributes; atIdx++) // start at 1 so we don't compare the id of each city
		{
			sum += ((dataset[instance1 + atIdx] - dataset[instance2 + atIdx]) * (dataset[instance1 + atIdx] - dataset[instance2 + atIdx]));
		}
		distances[row * numInstances + column] = (float) sqrt(sum);
		distances[column * numInstances + row] = distances[row * numInstances + column]; //set the distance for the other half of the pair we just computed
	}
}

__device__ unsigned long long countNumBits(unsigned long long n)
{
	unsigned long long count = 0;
	while (n)
	{
		count += n & 1;
		n >>= 1;
	}
	return count;
}

__device__ unsigned long long curPosition = 0;
__global__ void findPermutations(char* permutationsOfK, int k, unsigned long long lowerBound, unsigned long long upperBound)
{
	curPosition = 0;
	unsigned long long tid = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned long long numToCheck = lowerBound + tid;
	unsigned long long count = 0;
	unsigned long long curBitPosition = 0;
	if (numToCheck < upperBound)
	{

		if (countNumBits(numToCheck) == k)
		{
			__syncthreads();
			unsigned long long added = atomicAdd(&curPosition, 1);
			if (k == 1)
			{
				printf("found a permutation %llu\n", added);
			}
			unsigned long long permutationStartPos = (added) * (unsigned long long) k;
			while (numToCheck)
			{
				if (numToCheck & 1)
				{
					permutationsOfK[permutationStartPos + count] = curBitPosition;
					count++;
				}
				numToCheck >>= 1;
				curBitPosition++;
			}

		}
	}
}

vector<City> tsp(vector<City> cities, int numCities, float* distances, float* d_distances)
{

	hipEvent_t permutationsStart, permutationsStop;
	hipEventCreate(&permutationsStart);
	hipEventCreate(&permutationsStop);
	float permutationMilliseconds = 0;
	long long key = 0x00000;
	map<long long int, PathCost> solutionsMap;
	vector<int> cityNums;
	// convert cities back to integer array
	for (int i = 1; i < numCities; i++)
	{
		cityNums.push_back(i);
	}
	// calculate the highest layer number so we know how large we need to be for our permutation storage at worst
	int k = numCities % 2 == 0 ? numCities / 2 : (ceil(numCities / 2));
	// initalize first 2 levels of the lookup table
	for (int i = 1; i < numCities; i++)
	{
		for (int j = 1; j < numCities; j++)
		{
			if (i == j)
				continue;
			vector<int> iSet
			{ i };
			genKey(iSet, j, key);
			PathCost pathCost;
			vector<int> path
			{ 0, i };
			pathCost.path = path;
			pathCost.cost = distances[i * numCities + j] + distances[0 + i];
			solutionsMap.insert(pair<long long, PathCost>(key, pathCost));
		}
	}
	double currentCost = 0;
	char* d_permutationsOfK;
	char *h_permutationsOfK = (char*) malloc(pow(2, numCities) * sizeof(char) * k);
	gpuErrchk(hipMalloc(&d_permutationsOfK, pow(2, numCities) * sizeof(char) * k));

	unsigned long long finalPos;
	unsigned long long numPossibilities = pow(2, numCities); // - pow(2, k - 1);
	int threadsPerBlock = 1024;
	unsigned long long blocksPerGrid = ((numPossibilities) + threadsPerBlock - 1) / threadsPerBlock;
	for (int subsetSize = 2; subsetSize < numCities; subsetSize++)
	{
		hipEventRecord(permutationsStart);

		findPermutations<<<blocksPerGrid, threadsPerBlock, 0>>>(d_permutationsOfK, subsetSize, (unsigned long long) (pow(2, subsetSize) - 1),
				(unsigned long long) pow(2, numCities));
//		cudaDeviceSynchronize();
		gpuErrchk(hipMemcpyFromSymbol(&finalPos, HIP_SYMBOL(curPosition), sizeof(unsigned long long), 0, hipMemcpyDeviceToHost));
		gpuErrchk(hipMemcpy(h_permutationsOfK, d_permutationsOfK, finalPos * sizeof(char) * subsetSize, hipMemcpyDeviceToHost));

		hipEventRecord(permutationsStop);
		hipEventSynchronize(permutationsStop);
		hipEventElapsedTime(&permutationMilliseconds, permutationsStart, permutationsStop);
		printf("%i choose %i is %llu and took %f ms\n", numCities, subsetSize, finalPos, permutationMilliseconds);

		// use the permutations we generated here
		// remember the permutations are stored in k length 'arrays' within the 1-D array we have them in
		// so we need to index them interestingly.
		// converting to vector<vector<int>> so I don't have to rethink the logic at the current moment... definitely need to in the interests of speed
		vector<vector<int>> subsets;
		for (int pos = 0; pos < finalPos; pos++)
		{
			vector<int> permutation;
			for (int l = 0; l < subsetSize; l++)
			{
				permutation.push_back(h_permutationsOfK[pos * subsetSize + l]);
			}
			subsets.push_back(permutation);
		}
		int counter = 0;
		for (vector<int> set : subsets)
		{

			for (int k : set)
			{
				vector<int> kSet
				{ k };
				vector<int> diff;
				set_difference(set.begin(), set.end(), kSet.begin(), kSet.end(), inserter(diff, diff.begin()));
				double minCost = INT_MAX;
				vector<int> minPath;
				int bestM;
				counter++;
				// we initialized 2 levels earlier so this for loop will always be able to run.
				for (int m : diff)
				{
					vector<int> mSet
					{ m }; // need to generate the key for k-1
					vector<int> noMoreM; // get rid of m because thats where we're going
					set_difference(diff.begin(), diff.end(), mSet.begin(), mSet.end(), inserter(noMoreM, noMoreM.begin()));

					genKey(noMoreM, m, key);
					currentCost = solutionsMap[key].cost + distances[m * numCities + k];
					if (currentCost < minCost)
					{
						minCost = currentCost;
						minPath = solutionsMap[key].path;
						bestM = m;
					}
				}
				genKey(diff, k, key);

				PathCost pathCost;
				pathCost.cost = minCost;
				minPath.push_back(bestM);
				pathCost.path = minPath;
				solutionsMap.insert(pair<long long, PathCost>(key, pathCost));
			}
		}
		// printf("we have %i subsets of size %i\n", counter, i);
	}
	double minCost = INT_MAX;
	vector<int> minPath;
	int bestM;
	for (int m : cityNums)
	{
		vector<int> mSet
		{ m }; // need to generate the key for k-1
		vector<int> noMoreM; // get rid of m because thats where we're going
		set_difference(cityNums.begin(), cityNums.end(), mSet.begin(), mSet.end(), inserter(noMoreM, noMoreM.begin()));

		genKey(noMoreM, m, key);
		currentCost = solutionsMap[key].cost + distances[m * numCities + 0];
		if (currentCost < minCost)
		{
			minCost = currentCost;
			vector<int> path = solutionsMap[key].path;
			minPath = path;
			bestM = m;
		}
	}

	minPath.push_back(bestM);
	minPath.push_back(0);
	vector<City> bestPath;
	for (int i = 0; i < minPath.size(); i++)
	{
		bestPath.push_back(cities[minPath[i]]);
	}
	printf("Cost for this set of %i cities was %f\n", numCities, minCost);
	return bestPath;
}

int main(void)
{
	float* d_distances;
	float* h_distances;
	float* h_dataset;
	float* d_dataset;

	int numCities = 13;
	int numFeatures = 3;
	int k = numCities % 2 == 0 ? numCities / 2 : (ceil(numCities / 2));

	hipEvent_t allStart, allStop, distStart, distStop;
	hipEventCreate(&allStart);
	hipEventCreate(&allStop);
	hipEventCreate(&distStart);
	hipEventCreate(&distStop);

	float allMilliseconds = 0, distMilliseconds = 0;

	vector<City> cities = generateCities(numCities, 500, 500);

	hipHostMalloc(&h_dataset, sizeof(float) * numCities * numFeatures, hipHostMallocDefault);
	hipMalloc(&d_dataset, sizeof(float) * numCities * numFeatures);

	hipHostMalloc(&h_distances, sizeof(float) * numCities * numCities, hipHostMallocDefault);
	hipMalloc(&d_distances, sizeof(float) * numCities * numCities);

	for (int i = 0; i < numCities; i++) // convert cities vector to the array the distance computation kernel expects
	{
		h_dataset[i * numFeatures] = cities[i].id; //cities[i].id;
		h_dataset[i * numFeatures + 1] = cities[i].x;
		h_dataset[i * numFeatures + 2] = cities[i].y;
	}

	hipEventRecord(allStart);

	int threadsPerBlock = 1024;
	int blocksPerGrid = ((numCities * numCities) + threadsPerBlock - 1) / threadsPerBlock;

	hipEventRecord(distStart);
	gpuErrchk(hipMemcpy(d_dataset, h_dataset, numCities * numFeatures * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_distances, h_distances, numCities * numCities * sizeof(float), hipMemcpyHostToDevice));
	computeDistances<<<blocksPerGrid, threadsPerBlock, 0>>>(numCities, numFeatures, d_dataset, d_distances);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipMemcpy(h_distances, d_distances, numCities * numCities * sizeof(float), hipMemcpyDeviceToHost));
	hipEventRecord(distStop);
	hipEventSynchronize(distStop);

	gpuErrchk(hipFree(d_dataset));

	vector<City> solution = tsp(cities, numCities, h_distances, d_distances);

//	threadsPerBlock = 1024;
//	int numPossibilities = pow(2, numCities); // - pow(2, k - 1);
//	blocksPerGrid = ((numPossibilities) + threadsPerBlock - 1) / threadsPerBlock;
//
//	gpuErrchk(cudaFree(d_dataset));
//	cudaEventRecord(permutationsStart);
//	gpuErrchk(cudaMalloc(&d_permutationsOfK, pow(2, numCities) * sizeof(char) * k));
//	char *h_permutationsOfK = (char*) malloc(pow(2, 29) * sizeof(char) * k);
//
//	for (int i = 1; i <= numCities; i++)
//	{
//		findPermutations<<<blocksPerGrid, threadsPerBlock, 0>>>(d_permutationsOfK, i, (unsigned long long) (pow(2, i) - 1),
//				(unsigned long long) pow(2, numCities));
//		unsigned long long finalPos;
//		cudaDeviceSynchronize();
//		gpuErrchk(cudaMemcpyFromSymbol(&finalPos, curPosition, sizeof(unsigned long long), 0, cudaMemcpyDeviceToHost));
////		finalPos++;
//		gpuErrchk(cudaMemcpy(h_permutationsOfK, d_permutationsOfK, finalPos * sizeof(char) * i, cudaMemcpyDeviceToHost));
//
//		printf("%i choose %i is %llu\n", numCities, i, finalPos);
////		printf("permutations for size %i\n", i);
////		for (int j = 0; j < finalPos; j++)
////		{
////			for (int z = 0; z < i; z++)
////			{
////				printf("%i\t", (int) h_permutationsOfK[j * i + z]);
////			}
////			printf("\n");
////		}
//
//	}
	gpuErrchk(hipPeekAtLastError());
//	cudaDeviceSynchronize();

	hipEventRecord(allStop);
	hipEventSynchronize(allStop);
	hipEventElapsedTime(&distMilliseconds, distStart, distStop);

	hipEventElapsedTime(&allMilliseconds, allStart, allStop);

	printf("The distance calculation for %i cities took %llu ms.\n", numCities, (long long unsigned int) distMilliseconds);
//	printf("The permutations calculation for %i cities took %llu ms.\n", numCities, (long long unsigned int) permutationMilliseconds);
	printf("The salesman traversed %i cities in %llu ms.\n", numCities, (long long unsigned int) allMilliseconds);

	hipHostFree(h_dataset);
	hipHostFree(h_distances);
	hipFree(d_distances);
	return 0;
}

void printDistanceMatrix(float*h_distances, int numCities, int numFeatures)
{
	for (int i = 0; i < numCities; i++)
	{
		int city1Offset = i * numCities;
		for (int j = 0; j < numCities; j++)
		{
			printf("%f ", h_distances[city1Offset + j]);
		}
		printf("\n");
	}
}

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
// https://stackoverflow.com/a/14038590
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort)
			exit(code);
	}
}

double fRand(double fMin, double fMax)
{
	double f = (double) rand() / RAND_MAX;
	return fMin + f * (fMax - fMin);
}

vector<City> generateCities(int numCities, int gridDimX, int gridDimY)
{
	vector<City> cities;
	for (int i = 0; i < numCities; i++)
	{
		City city;
		city.id = i;
		city.x = fRand(0, gridDimX);
		city.y = fRand(0, gridDimY);
		cities.push_back(city);
	}
//	City city0;
//	city0.id = 0;
//	city0.x = 323.05;
//	city0.y = 24.73;
//	cities.push_back(city0);
//	City city1;
//	city1.id = 1;
//	city1.x = 24.56;
//	city1.y = 101.00;
//	cities.push_back(city1);
//	City city2;
//	city2.id = 2;
//	city2.x = 275.87;
//	city2.y = 44.57;
//	cities.push_back(city2);
//	City city3;
//	city3.id = 3;
//	city3.x = 114.67;
//	city3.y = 186.45;
//	cities.push_back(city3);
//	City city4;
//	city4.id = 4;
//	city4.x = 164.11;
//	city4.y = 334.44;
//	cities.push_back(city4);
//	City city5;
//	city5.id = 5;
//	city5.x = 485.90;
//	city5.y = 401.21;
//	cities.push_back(city5);
//	City city6;
//	city6.id = 6;
//	city6.x = 333.49;
//	city6.y = 464.63;
//	cities.push_back(city6);
//	City city7;
//	city7.id = 7;
//	city7.x = 133.37;
//	city7.y = 168.05;
//	cities.push_back(city7);
//	City city8;
//	city8.id = 8;
//	city8.x = 362.79;
//	city8.y = 255.52;
//	cities.push_back(city8);
//	City city9;
//	city9.id = 9;
//	city9.x = 378.74;
//	city9.y = 235.48;
//	cities.push_back(city9);

	return cities;
}

void genKey(vector<int> set, int z, long long &key)
{
	key = 0;
	key |= z;
	for (int j : set)
	{
		key |= (1 << (j + 8));
	}
}
